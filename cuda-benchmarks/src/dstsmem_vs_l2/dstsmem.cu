#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <helper_string.h>
#include <cuda_error_handling.h>
#include "dstsmem_vs_l2.h"

namespace cg = cooperative_groups;

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 1024
#endif

__global__ void kernel(const int* input, const int input_size, int* output)
{
    __shared__ int smem[BLOCK_SIZE];

    cg::grid_group grid = cg::this_grid();
    cg::thread_block block = cg::this_thread_block();
    cg::cluster_group cluster = cg::this_cluster();

    unsigned tbc_size = cluster.num_blocks();
    unsigned cbr = cluster.block_rank();

    for (int i = 0; i <= input_size / BLOCK_SIZE; i++)
    {
        int index = block.thread_rank() 
                  + i * BLOCK_SIZE * tbc_size
                  + cbr * BLOCK_SIZE;
        smem[block.thread_rank()]
            = index < input_size ? input[index] : 0;

        cg::sync(cluster);

        const int* dst_smem = cluster.map_shared_rank(smem, 0);
        int value = dst_smem[0];
        
        for (int l = 0; l < tbc_size; l++)
        {
            dst_smem = cluster.map_shared_rank(smem, l);
            for (int j = 1; j < BLOCK_SIZE; j++)
                value = dst_smem[j] < value ? dst_smem[j] : value;
        }
        
        if (block.thread_rank() == 0)
            output[grid.block_rank() + i * grid.num_blocks()] = value; 

        cg::sync(cluster);
    }
}

int main(int argc, char* argv[])
{
    int input_size = 1024;
    if (checkCmdLineFlag(argc, (const char**) argv, "Input_Size"))
        input_size = getCmdLineArgumentInt(argc, (const char**) argv, "Input_Size");

    int num_blocks = 64;
    if (checkCmdLineFlag(argc, (const char**) argv, "Number_of_Blocks"))
        num_blocks = getCmdLineArgumentInt(argc, (const char**) argv, "Number_of_Blocks");

    int tbc_size = 8;
    if (checkCmdLineFlag(argc, (const char**) argv, "Blocks_per_Cluster"))
        tbc_size = getCmdLineArgumentInt(argc, (const char**) argv, "Blocks_per_Cluster");

    if (num_blocks % tbc_size)
    {
        std::cout << "Number of blocks is not divisible by TBC-Size." << std::endl;
        exit(EXIT_FAILURE);
    }

    int output_size = num_blocks * ((input_size / BLOCK_SIZE) + 1);
    
    int* input = (int*) malloc(input_size * sizeof(int));
    int* output = (int*) malloc(output_size * sizeof(int));
    int* d_input = nullptr;
    int* d_output = nullptr;

    CUDA_CHECK(hipMalloc((void**) &d_input, input_size * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**) &d_output, output_size * sizeof(int)));

    init(input, input_size, output, output_size);
    CUDA_CHECK(hipMemcpy(d_input, input, input_size * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_output, output, output_size * sizeof(int), hipMemcpyHostToDevice));

    // Setting launch attributes
    hipLaunchConfig_t config = {0};
    config.gridDim.x = num_blocks;
    config.blockDim.x = BLOCK_SIZE;

    std::cout << "=== Launch parameters ===" << std::endl;
    std::cout << "Blocks: " << num_blocks << std::endl;
    std::cout << "Input size: " << input_size << std::endl;
    std::cout << "Output size: " << output_size << std::endl;
    std::cout << "Threads per block: " << config.blockDim.x << std::endl;
    std::cout << "Blocks per cluster: " << tbc_size << std::endl;

    CUDA_CHECK(cudaLaunchKernelEx(&config, kernel, d_input, input_size, d_output));
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(output, d_output, output_size * sizeof(int), hipMemcpyDeviceToHost));

    std::cout << "=== Results ===" << std::endl;
    verify_result(output, output_size, num_blocks);

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    free(input);
    free(output);
}
