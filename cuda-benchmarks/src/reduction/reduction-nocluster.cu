#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>

#include "large_block.cuh"
#include "cuda_error_handling.h"
#include "helper_string.h"

#include "reduction.h"

namespace cg = cooperative_groups;

__global__ void Reduction(int* input, const int length, const int iterations, int* output)
{
    extern __shared__ int smem[];

    cg::thread_block block = cg::this_thread_block();  
    cg::grid_group grid = cg::this_grid();

    int grid_size = grid.size();
    int block_size = block.size();
    
    int result = 0;
    int index = (int) grid.thread_rank();

    // Sum array with index difference of grid size.
    for (int i = 0; i < iterations; i++)
    {
        if (index < length)
            result += input[index];
        
        index += grid_size;
    }
    
    // Load into shared memory.
    unsigned btr = block.thread_rank();
    smem[btr] = result;
    cg::sync(block);

    // Sum block.
    for (int i = block_size / 2; i > 0; i = i / 2)
    {
        if (btr < i)
            smem[btr] += smem[btr + i];
        cg::sync(block);
    }
    
    // Add final values.
    if (btr == 0)
        atomicAdd(output, smem[0]);
}

int main(int argc, char* argv[])
{
    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));

    if (checkCmdLineFlag(argc, (const char**) argv, "help"))
    {
        std::cout << "reduction: \n"
                  << "--Number_of_Blocks\n"
                  << "--Array_Size\n"
                  << "--Threads_per_Block\n";
        exit(0);
    }

    // Initialisation
    int blocks = 64;
    int input_size = 1000000;
    int threads_per_block = 1024;

    if (checkCmdLineFlag(argc, (const char**) argv, "Threads_per_Block"))
        threads_per_block = getCmdLineArgumentInt(argc, (const char**) argv, "Threads_per_Block");

    int smem_size = threads_per_block * sizeof(int);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&blocks, Reduction, threads_per_block, smem_size);
    blocks *= deviceProp.multiProcessorCount;

    if (checkCmdLineFlag(argc, (const char**) argv, "Number_of_Blocks"))
        blocks = getCmdLineArgumentInt(argc, (const char**) argv, "Number_of_Blocks");
    
    if (checkCmdLineFlag(argc, (const char**) argv, "Array_Size"))
        input_size = getCmdLineArgumentInt(argc, (const char**) argv, "Array_Size");

    // Allocate memory, init variables and calculate cpu result.
    int* input = (int*) malloc(sizeof(int) * input_size);
    int* d_input = nullptr;
    int* d_output = nullptr;

    init_array_random(input, input_size);
    int cpu_result = reduction_cpu(input, input_size);

    CUDA_CHECK(hipMalloc((void**) &d_input, input_size * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**) &d_output, sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_input, input, input_size * sizeof(int), hipMemcpyHostToDevice));

    int iterations = (input_size / (threads_per_block * blocks)) + 1;

    std::cout << "Blocks: " << blocks << "\n"
              << "Threads per Block: " << threads_per_block << "\n"
              << "Shared Memory size: " << smem_size << "\n"
              << "Iterations: " << iterations << "\n"
              << "Length: " << input_size << "\n";

    // CUDA kernal call.
    Reduction<<<blocks, threads_per_block, smem_size>>>(d_input, input_size, iterations, d_output);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy back memory and compare results.
    int cuda_result = 0;
    CUDA_CHECK(hipMemcpy(&cuda_result, d_output, sizeof(int), hipMemcpyDeviceToHost));

    cmp_results(&cpu_result, &cuda_result, 1);
    
    // Free ressources.
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_input));
    free(input);
}